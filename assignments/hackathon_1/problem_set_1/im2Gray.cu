#include "hip/hip_runtime.h"
#include "im2Gray.h"
#include <math.h> 

//#define BLOCK 32

/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols, size_t block_size){
  // Create shared memory uchar4 to hold tile data of size equal to block_size
  __shared__ uchar4 pixels[block_size][block_size];
  // Create shared memory unsigned char array to hold grey outputs
  __shared__ unsigned char grey_pixels[block_size][block_size];

  // Get location of pixel in global memory
  int gl_row = blockIdx.y * blockDim.y + threadIdx.y;
  int gl_col = blockIdx.x * blockDim.x + threadIdx.x;

  // Get location of pixel in shared memory
  int sh_row = threadIdx.y;
  int sh_col = threadIdx.x;

  // Load shared memory from global memory
  if (gl_col < numCols && gl_row < numRows){
    int global_offset = gl_row * numCols + gl_col;
    pixels[sh_row][sh_col] = d_in[global_offset];
  }

  // Make sure all threads have loaded before starting computation
  __syncthreads();

  // Compute Grey Pixel and store results in shared memory
  if (gl_col < numCols && gl_row < numRows){
    uchar4 rgba_pixel = pixels[sh_row][sh_col];
    grey_pixels[sh_row][sh_col] = (unsigned char)((float)rgba_pixel.x*0.299f + (float)rgba_pixel.y*0.587f + (float)rgba_pixel.z*0.114f);
  }

  // Make sure all threads have finished computation
  __syncthreads();

  // Write all results back to global memory output array
  if (gl_col < numCols && gl_row < numRows){
    int grey_offset = gl_row * numCols + gl_col;
    d_grey[grey_offset] = grey_pixels[sh_row][sh_col];
  }
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols, size_t block_size){
    // Ensure block_size is valid
    if (block_size > 32){
      block_size = 32;
    } else if (block_size <= 0){
      block_size = 1;
    }
    // Set grid and block dimensions
    dim3 grid(std::ceil((float)numCols/(float)block_size),std::ceil((float)numRows/(float)block_size),1);
    dim3 block(block_size, block_size, 1);

    // Call Kernel
    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols, block_size);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}





