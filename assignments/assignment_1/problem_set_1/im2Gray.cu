#include "hip/hip_runtime.h"
#include "im2Gray.h"
#include <math.h> 

#define BLOCK 32

/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){
  // Column Indicator
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  // Row Indicator
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Only do valid pixel locations
  if(x < numCols && y < numRows) {
    // Get one dimension array offset
    int grey_Offset = y * numCols + x;
    // Get corresponding rgba pixel at this location
    uchar4 rgba_pixel = d_in[grey_Offset];
    // Calculate new grey value
    d_grey[grey_Offset] = (unsigned char)((float)rgba_pixel.x*0.299f + (float)rgba_pixel.y*0.587f + (float)rgba_pixel.z*0.114f);
  }
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // Ensure there are not over BLOCK number of blocks
    // Given the number of total blocks, determine the number of threads needed per block

    // Configuration 1
    //dim3 grid(numCols,numRows,1); 
    //dim3 block(1,1,1); 


    // Configuration 2
    /**
    size_t grid_x = std::ceil((float)BLOCK/2);
    size_t grid_y = std::ceil((float)BLOCK/2);
    size_t block_x = std::ceil((float)numCols/grid_x);
    size_t block_y = std::ceil((float)numRows/grid_y);
    size_t block_size = block_x * block_y;
    size_t new_block = BLOCK;

    while (block_size > 1024){
      new_block = new_block * 2;
      grid_x = std::ceil((float)new_block/2);
      grid_y = std::ceil((float)new_block/2);
      block_x = std::ceil((float)numCols/grid_x);
      block_y = std::ceil((float)numRows/grid_y);
      block_size = block_x * block_y;
    } 

    std::cout << "x: " << grid_x << " y: " << grid_y << std::endl;
    dim3 grid(grid_x,grid_y,1); 
    std::cout << "x2: " << block_x << " y2: " << block_y << std::endl;
    dim3 block(block_x,block_y,1);  
    **/

    // Configuration 3
    dim3 grid(1,numRows,1);
    dim3 block(numCols,1,1); 

    // Call Kernel
    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}





